#include <utilities/base_fixture.hpp>
#include <utilities/test_utilities.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/legacy/graph.hpp>

#include <raft/error.hpp>
#include <raft/handle.hpp>

#include <hip/hip_runtime_api.h>

#include <cmath>

#include "../src/converters/legacy/COOtoCSR.cuh"
#include <thrust/device_ptr.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/reduce.h>

#include <sys/time.h>

int main(int argc, char* argv[]) {

  if (argc != 2) {
    printf("USAGE: %s input_graph\n", argv[0]);
    exit(-1);
  }

  printf("%s\n", argv[1]); //input

  FILE* fpin = fopen(argv[1], "r");

  int m, k, nnz;
  MM_typecode mc;

  cugraph::test::mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz);

  // Allocate memory on host
  std::vector<int> cooRowInd(nnz), cooColInd(nnz);
  std::vector<double> cooVal(nnz), mst(m);

  // Read
  cugraph::test::mm_to_coo<int, double>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL);
  fclose(fpin);

  raft::handle_t handle;

  cugraph::legacy::GraphCOOView<int, int, double> G_coo(
      &cooRowInd[0], &cooColInd[0], &cooVal[0], m, nnz);
  auto G_unique = cugraph::coo_to_csr(G_coo);
  cugraph::legacy::GraphCSRView<int, int, double> G(G_unique->view().offsets,
                                                   G_unique->view().indices,
                                                   G_unique->view().edge_data,
                                                   G_unique->view().number_of_vertices,
                                                   G_unique->view().number_of_edges);

  hipDeviceSynchronize();

  auto mst_edges1 = cugraph::minimum_spanning_tree<int, int, double>(handle, G);
  hipDeviceSynchronize();

  timeval start, end;
  gettimeofday(&start, NULL);

  auto mst_edges = cugraph::minimum_spanning_tree<int, int, double>(handle, G);
  hipDeviceSynchronize();

  gettimeofday(&end, NULL);
  printf("%12.9f s\n", end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0);

  auto expected_mst_weight = thrust::reduce(
      thrust::device_pointer_cast(G_unique->view().edge_data),
      thrust::device_pointer_cast(G_unique->view().edge_data) + G_unique->view().number_of_edges);

  auto calculated_mst_weight = thrust::reduce(
      thrust::device_pointer_cast(mst_edges->view().edge_data),
      thrust::device_pointer_cast(mst_edges->view().edge_data) + mst_edges->view().number_of_edges);

  printf("calc weight: %ld\n", (long)calculated_mst_weight);
  printf("calc weight: %lx\n", (long)calculated_mst_weight);
