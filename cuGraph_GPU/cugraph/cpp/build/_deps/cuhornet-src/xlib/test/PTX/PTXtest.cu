#include "hip/hip_runtime.h"
#include "XLib.hpp"
using namespace timer;

struct DEV_CONST {
    int* __restrict__ ptr;
    int* __restrict__ ptr1;
    int* __restrict__ ptr2;
    int* __restrict__ ptr3;
};

__constant__ DEV_CONST dev_const;

__constant__ int* const_ptr;

__device__ __forceinline__ void write() {
    for (int i = 0; i < 100; i++)
        dev_const.ptr[i] = dev_const.ptr3[i] ;
}


__global__ void use_ptr() {
    write();
}

__global__ void use_ptr2(int* __restrict__ dev_const) {
    for (int i = 0; i < 100; i++)
        dev_const[i] = i;
}

__global__ void use_ptr3() {
    for (int i = 0; i < 100; i++)
        const_ptr[i] = i;
}

__device__ int Array[256];

__global__ void PTXoperation() {
    Array[0] = (int) threadIdx.x % 16;
    Array[1] = (int) threadIdx.x % 16u;
    Array[2] = (int) threadIdx.x & 15;

    Array[3] = (int) threadIdx.x / 16;
    Array[4] = (int) threadIdx.x / 16u;
    Array[5] = (int) threadIdx.x >> 4;
}

__device__ int2 devInt2[10];

__global__ void regOperation() {
    int2 f = devInt2[1];
    devInt2[0] = f;
}


__global__ void regOperation2() {
    int2 f = devInt2[1];
    devInt2[0].x = f.x;
    devInt2[0].y = f.y;
}

const int SIZE = 1 << 22;
__device__ int ArrayIN[SIZE];
__device__ int ArrayOUT[SIZE];

template<unsigned SPLIT = 1>
__global__ void storeKernel() {
    int* ptrOUT = ArrayOUT;
    int end = SIZE;

    if (SPLIT != 1) {
        ptrOUT += (threadIdx.x / (32 / SPLIT)) * (SIZE / SPLIT);
        end = ((threadIdx.x / (32 / SPLIT)) + 1) * (SIZE / SPLIT);
    }

    for (int i = threadIdx.x % (32 / SPLIT); i < end; i += 32 / SPLIT)
        ArrayOUT[i] = 1;
}

int main() {
    int* ptr_host;
    hipMalloc(&ptr_host, 100 * sizeof(int));

    DEV_CONST dev_const_h;
    dev_const_h.ptr = ptr_host;
    hipMemcpyToSymbol(HIP_SYMBOL(dev_const), &dev_const_h, sizeof(DEV_CONST));


    CUDA_ERROR("dd");

    PTXoperation<<<1, 1>>>();
    Timer_cuda TM;
    TM.start();

    storeKernel<<<1, 32>>>();

    TM.getTime();
    TM.start();

    storeKernel<2><<<1, 32>>>();

    TM.getTime("split = 16");
}
