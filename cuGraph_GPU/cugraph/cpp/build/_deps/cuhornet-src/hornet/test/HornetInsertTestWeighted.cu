#include "Hornet.hpp"
#include "StandardAPI.hpp"
#include "Util/BatchFunctions.hpp"
#include "Util/RandomGraphData.cuh"
#include <Host/FileUtil.hpp>            //xlib::extract_filepath_noextension
#include <Device/Util/CudaUtil.cuh>     //xlib::deviceInfo
#include <algorithm>                    //std:.generate
#include <chrono>                       //std::chrono
#include <random>                       //std::mt19937_64
#include <hip/hip_runtime_api.h>

#include <Graph/GraphStd.hpp>
#include <Host/Classes/Timer.hpp>
#include <Device/Util/Timer.cuh>
#include "Util/CommandLineParam.hpp"

//using namespace hornets_nest;
using namespace timer;
using namespace std::string_literals;

using vert_t = int;
using eoff_t = int;
using wgt0_t = int;
using wgt1_t = float;
using Init = hornet::HornetInit<vert_t, hornet::EMPTY, hornet::TypeList<wgt0_t, wgt1_t>>;
using HornetGPU = hornet::gpu::Hornet<vert_t, hornet::EMPTY, hornet::TypeList<wgt0_t, wgt1_t>>;
using UpdatePtr = hornet::BatchUpdatePtr<vert_t, hornet::TypeList<wgt0_t, wgt1_t>, hornet::DeviceType::HOST>;
using Update = hornet::gpu::BatchUpdate<vert_t, hornet::TypeList<wgt0_t, wgt1_t>>;
using hornet::TypeList;
using hornet::DeviceType;

/**
 * @brief Example tester for Hornet
 */
int exec(int argc, char* argv[]) {
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    graph::GraphStd<vert_t, vert_t> graph;
    graph.read(argv[1]);
    int batch_size = std::stoi(argv[2]);
    Init hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(), graph.csr_out_edges());

    //Use meta with hornet_init
    std::vector<wgt0_t> edge_meta_0(graph.nE(), 0);
    std::vector<wgt1_t> edge_meta_1(graph.nE(), 1);
    hornet_init.insertEdgeData(edge_meta_0.data(), edge_meta_1.data());
    HornetGPU hornet_gpu(hornet_init);
    auto init_coo = hornet_gpu.getCOO(true);

    hornet::RandomGenTraits<TypeList<wgt0_t, wgt1_t>> cooGenTraits;
    auto randomBatch = hornet::generateRandomCOO<vert_t, eoff_t>(graph.nV(), batch_size, cooGenTraits);
    Update batch_update(randomBatch);

    printf("ne: %d\n", hornet_gpu.nE());
    std::cout<<"=======\n";
    Timer<DEVICE> TM(3);
    TM.start();
    hornet_gpu.insert(batch_update);

    TM.stop();

    printf("ne: %d\n", hornet_gpu.nE());
    std::cout<<"=======\n";
    TM.print("Insertion " + std::to_string(batch_size) + ":  ");

    auto inst_coo = hornet_gpu.getCOO(true);
    init_coo.append(randomBatch);
    init_coo.sort();

    std::cout<<"Creating multimap for testing correctness...";
    auto init_coo_map = getHostMMap(init_coo);
    auto inst_coo_map = getHostMMap(inst_coo);
    std::cout<<"...Done!\n";
    if (inst_coo_map == inst_coo_map) {
      std::cout<<"Passed\n";
    } else {
      std::cout<<"Failed\n";
    }

    return 0;
}

int main(int argc, char* argv[]) {
  int ret = 0;
  {
    ret = exec(argc, argv);
  }

  return ret;
}
