/**
 * @brief Breadth-first Search Top-Down test program
 * @file
 */
#include "Static/BreadthFirstSearch/TopDown2.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

template <typename HornetGraph, typename BFS>
int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;

    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv,false);


    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    Timer<DEVICE> TM;
    hipProfilerStart();
    TM.start();
    HornetGraph hornet_graph(hornet_init);
    TM.stop();
    hipProfilerStop();
    TM.print("Initilization Time:");

    BFS bfs_top_down(hornet_graph);

    vid_t root = graph.max_out_degree_id();
    if (argc==3)
        root = atoi(argv[2]);

    std::cout << "My root is " << root << std::endl;

    bfs_top_down.set_parameters(root);

    hipProfilerStart();
    TM.start();

    bfs_top_down.run();

    TM.stop();
    hipProfilerStop();
    TM.print("TopDown2");

    std::cout << "Number of levels is : " << bfs_top_down.getLevels() << std::endl;

    auto is_correct = bfs_top_down.validate();
    std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
    return !is_correct;
}

int main(int argc, char* argv[]) {
  int ret = 0;
  {
    ret = exec<hornets_nest::HornetStaticGraph,  hornets_nest::BfsTopDown2Static >(argc, argv);
  }

  return ret;
}
