/**
 * @internal
 * @author Oded Green                                                  <br>
 *         Georgia Institute of Technology, Computational Science and Engineering                   <br>
 *         ogreen@gatech.edu
 * @date August, 2017
 * @version v2
 *
 * @copyright Copyright © 2017 cuStinger. All rights reserved.
 *
 * @license{<blockquote>
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * * Neither the name of the copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * </blockquote>}
 *
 * @file
 */
#include "Dynamic/KatzCentrality/Katz.cuh"
#include <StandardAPI.hpp>
#include <Device/Util/Timer.cuh>
#include <Graph/GraphStd.hpp>

int exec(int argc, char* argv[]) {
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;
    using namespace graph;
    using namespace hornets_nest;
    using namespace timer;

    int max_iterations = 1000;
    int           topK = 100;

    GraphStd<vid_t, eoff_t> graph(UNDIRECTED | ENABLE_INGOING);
    graph.read(argv[1], SORT | PRINT_INFO);

    HornetInit hornet_init(graph.nV(), graph.nE(),
                           graph.csr_out_offsets(),
                           graph.csr_out_edges());

	HornetGraph hornet_graph(hornet_init);

    HornetInit hornet_init_inverse(graph.nV(), graph.nE(),
                                   graph.csr_in_offsets(),
                                   graph.csr_in_edges());

    // Finding largest vertex
    degree_t max_degree_vertex = hornet_graph.max_degree_id();
    std::cout << "Max degree vextex is " << max_degree_vertex << std::endl;

	KatzCentrality kcPostUpdate(hornet_graph, max_iterations, topK,
                                max_degree_vertex);
	Timer<DEVICE> TM;
	TM.start();

	kcPostUpdate.run();

	TM.stop();
    std::cout<<"Dynamic Katz\n";
    auto total_time = TM.duration();
    std::cout << "The number of iterations   : "
              << kcPostUpdate.get_iteration_count()
              << "\nTotal time for KC          : " << total_time
              << "\nAverage time per iteartion : "
              << total_time /
                 static_cast<float>(kcPostUpdate.get_iteration_count())
              << "\n";

    return 0;
}

int main(int argc, char* argv[]) {
  int ret = 0;
  {

    ret = exec(argc, argv);

  }

  return ret;
}
