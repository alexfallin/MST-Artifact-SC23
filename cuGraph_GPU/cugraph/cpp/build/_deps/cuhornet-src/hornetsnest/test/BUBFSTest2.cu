/**
 * @brief Breadth-first Search Top-Down test program
 * @file
 */
#include "Static/BUBreadthFirstSearch/BottomUpBFS.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;
    using vid_t = int;
    using dst_t = int;

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    // graph::GraphStd<vid_t, eoff_t> graph;
    graph::GraphStd<vid_t, eoff_t> graph(DIRECTED | ENABLE_INGOING);
    CommandLineParam cmd(graph, argc, argv,false);


    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());


    HornetInit hornet_init_inverse(graph.nV(), graph.nE(),
                                   graph.csr_in_offsets(),
                                   graph.csr_in_edges());



    HornetGraph hornet_graph_inv(hornet_init_inverse);
    HornetGraph hornet_graph(hornet_init);


    BfsBottomUp2 bfs_bottom_up(hornet_graph, hornet_graph_inv);
    BfsBottomUp2 bfs_top_down(hornet_graph, hornet_graph_inv);

	vid_t root = graph.max_out_degree_id();
	if (argc==3)
	  root = atoi(argv[2]);

    bfs_bottom_up.set_parameters(root);
    bfs_top_down.set_parameters(root);


    Timer<DEVICE> TM;
    hipProfilerStart();
    TM.start();

    //bfs_top_down.run();
    bfs_bottom_up.run(hornet_graph_inv);

    TM.stop();
    hipProfilerStop();
    TM.print("BottomUp2");

    auto is_correct = bfs_bottom_up.validate();
    std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
    return !is_correct;
}

int main(int argc, char* argv[]) {
  int ret = 0;
  {

    ret = exec(argc, argv);

  }

  return ret;
}
