/**
 * @brief
 * @author Oded Green                                                       <br>
 *   Georgia Institute of Technology, Computational Science and Engineering <br>                   <br>
 *   ogreen@gatech.edu
 * @date August, 2017
 * @version v2
 *
 * @copyright Copyright © 2017 Hornet. All rights reserved.
 *
 * @license{<blockquote>
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * * Neither the name of the copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * </blockquote>}
 *
 * @file
 */
#include "Static/KatzCentrality/KatzTopK.cuh"
#include <StandardAPI.hpp>
#include <Device/Util/Timer.cuh>
#include <Graph/GraphStd.hpp>

template <typename HornetGraph,typename Katz>
int exec(int argc, char* argv[]) {
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;
    using namespace graph;
    using namespace hornets_nest;
    using namespace timer;

	// Limit the number of iteartions for graphs with large number of vertices.
    int max_iterations = 50;

	  hipSetDevice(0);
    GraphStd<vert_t, vert_t> graph(UNDIRECTED);

    graph.read(argv[1], SORT | PRINT_INFO);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(), graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);
 	  // Users can add the number of TopK vertices for the approximation
	  int topK = graph.nV();
     if(argc>2)
        topK=atoi(argv[2]);

    // Finding largest vertex degreemake
    degree_t max_degree_vertex = hornet_graph.max_degree();
    std::cout << "Max degree vextex is " << max_degree_vertex << std::endl;


    Katz kcPostUpdate(hornet_graph, max_iterations, topK, max_degree_vertex);


    Timer<DEVICE> TM;
    TM.start();

    kcPostUpdate.run();

    TM.stop();

    auto total_time = TM.duration();
    std::cout << "The number of iterations     : "
              << kcPostUpdate.get_iteration_count()
              << "\nTopK                       : " << topK
              << "\nTotal time for KC          : " << total_time
              << "\nAverage time per iteartion : "
              << total_time /
                 static_cast<float>(kcPostUpdate.get_iteration_count())
              << "\n";

    return 0;
}

int main(int argc, char* argv[]) {
  int ret = 0;
  {

    for(int i=0; i<10; i++){
      ret = exec<hornets_nest::HornetDynamicGraph,hornets_nest::KatzCentralityTopKDynamicH>(argc, argv);
      ret = exec<hornets_nest::HornetStaticGraph,hornets_nest::KatzCentralityTopKStatic>(argc, argv);

    }

  }

  return ret;
}
