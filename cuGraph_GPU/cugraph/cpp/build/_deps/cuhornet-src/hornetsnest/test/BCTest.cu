/**
 * @brief Betweenness Centrality test program
 * @file
 */
#include "Static/BetweennessCentrality/bc.cuh"
#include "Static/BetweennessCentrality/exact_bc.cuh"
#include "Static/BetweennessCentrality/approximate_bc.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

using namespace std;
using namespace graph;
using namespace graph::structure_prop;
using namespace graph::parsing_prop;

int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;

    // GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    // graph::GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv,false);
    Timer<DEVICE> TM;


    // graph.read(argv[1], SORT | PRINT_INFO);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);

    BCCentrality bc(hornet_graph);

	vid_t root = graph.max_out_degree_id();
	if (argc==3)
	  root = atoi(argv[2]);
    // root = 226410;
    cout << "Root is " << root << endl;
    bc.reset();
    bc.setRoot(root);

    hipProfilerStart();TM.start();
    bc.run();

    TM.stop();hipProfilerStop();
    TM.print("BCCentrality");
#if 0
    // auto is_correct = bc.validate();
    // std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
    // return !is_correct;
    ExactBC ebc(hornet_graph);

    ebc.reset();

    hipProfilerStart();TM.start();
    // ebc.run();
    TM.stop();hipProfilerStop();
    TM.print("Exact BCCentrality");

    vid_t numRoots=1000;
    vid_t* roots = new vid_t[numRoots];
    ApproximateBC::generateRandomRootsUniform(hornet_graph.nV(), numRoots, &roots, 1 );

    ApproximateBC abc(hornet_graph, roots,numRoots);
    abc.reset();

    hipProfilerStart();TM.start();
    // abc.run();
    TM.stop();hipProfilerStop();
    TM.print("Approximate BCCentrality");


    delete[] roots;
#endif
    return 0;
}

int main(int argc, char* argv[]) {
  int ret = 0;
  {

    ret = exec(argc, argv);

  }

  return ret;
}
