#include "hip/hip_runtime.h"
/*
Please cite:
* J. Fox, O. Green, K. Gabert, X. An, D. Bader, “Fast and Adaptive List Intersections on the GPU”, 
IEEE High Performance Extreme Computing Conference (HPEC), 
Waltham, Massachusetts, 2018
* O. Green, J. Fox, A. Tripathy, A. Watkins, K. Gabert, E. Kim, X. An, K. Aatish, D. Bader, 
“Logarithmic Radix Binning and Vectorized Triangle Counting”, 
IEEE High Performance Extreme Computing Conference (HPEC), 
Waltham, Massachusetts, 2018
* O. Green, P. Yalamanchili ,L.M. Munguia, “Fast Triangle Counting on GPU”, 
Irregular Applications: Architectures and Algorithms (IA3), 
New Orleans, Louisiana, 2014 
*/



#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Static/TriangleCounting/triangle2.cuh"

namespace hornets_nest {

TriangleCounting2::TriangleCounting2(HornetGraph& hornet) :
                                       StaticAlgorithm(hornet)

{                                       
}

TriangleCounting2::~TriangleCounting2(){
    release();
}

struct OPERATOR_InitTriangleCounts {
    triangle_t *d_triPerVertex;

    OPERATOR (Vertex &vertex) {
        d_triPerVertex[vertex.id()] = 0;
    }
};

/*
 * Naive intersection operator
 * Assumption: access to entire adjacencies of v1 and v2 required
 */
struct OPERATOR_AdjIntersectionCount {
    triangle_t* d_triPerVertex;

    OPERATOR(Vertex& v1, Vertex& v2, int flag) {
        triangle_t count = 0;
        int deg1 = v1.degree();
        int deg2 = v2.degree();
        vid_t* ui_begin = v1.neighbor_ptr();
        vid_t* vi_begin = v2.neighbor_ptr();
        vid_t* ui_end = ui_begin+deg1-1;
        vid_t* vi_end = vi_begin+deg2-1;
        int comp_equals, comp1, comp2;
        while (vi_begin <= vi_end && ui_begin <= ui_end) {
            comp_equals = (*ui_begin == *vi_begin);
            count += comp_equals;
            comp1 = (*ui_begin >= *vi_begin);
            comp2 = (*ui_begin <= *vi_begin);
            vi_begin += comp1;
            ui_begin += comp2;
            // early termination
            if ((vi_begin > vi_end) || (ui_begin > ui_end))
                break;
        }
        atomicAdd(d_triPerVertex+v1.id(), count);
        atomicAdd(d_triPerVertex+v2.id(), count);
    }
};


struct OPERATOR_AdjIntersectionCountBalanced {
    triangle_t* d_triPerVertex;

    OPERATOR(Vertex &u, Vertex& v, vid_t* ui_begin, vid_t* ui_end, vid_t* vi_begin, vid_t* vi_end, int FLAG) {
        int count = 0;
        if (!FLAG) {
            int comp_equals, comp1, comp2, ui_bound, vi_bound;
            //printf("Intersecting %d, %d: %d -> %d, %d -> %d\n", u.id(), v.id(), *ui_begin, *ui_end, *vi_begin, *vi_end);
            while (vi_begin <= vi_end && ui_begin <= ui_end) {
                comp_equals = (*ui_begin == *vi_begin);
                count += comp_equals;
                comp1 = (*ui_begin >= *vi_begin);
                comp2 = (*ui_begin <= *vi_begin);
                ui_bound = (ui_begin == ui_end);
                vi_bound = (vi_begin == vi_end);
                // early termination
                if ((ui_bound && comp2) || (vi_bound && comp1))
                    break;
                if ((comp1 && !vi_bound) || ui_bound)
                    vi_begin += 1;
                if ((comp2 && !ui_bound) || vi_bound)
                    ui_begin += 1;
            }
        } else {
            vid_t vi_low, vi_high, vi_mid;
            while (ui_begin <= ui_end) {
                auto search_val = *ui_begin;
                vi_low = 0;
                vi_high = vi_end-vi_begin;
                while (vi_low <= vi_high) {
                    vi_mid = (vi_low+vi_high)/2;
                    auto comp = (*(vi_begin+vi_mid) - search_val);
                    if (!comp) {
                        count += 1;
                        break;
                    }
                    if (comp > 0) {
                        vi_high = vi_mid-1;
                    } else if (comp < 0) {
                        vi_low = vi_mid+1;
                    }
                }
                ui_begin += 1;
            }
        }

        atomicAdd(d_triPerVertex+u.id(), count);
        //atomicAdd(d_triPerVertex+v.id(), count);
    }
};

void TriangleCounting2::copyTCToHost(triangle_t* h_tcs) {
    gpu::copyToHost(triPerVertex, hornet.nV(), h_tcs);
}

triangle_t TriangleCounting2::countTriangles(){

    triangle_t* h_triPerVertex;
    host::allocate(h_triPerVertex, hornet.nV());
    gpu::copyToHost(triPerVertex, hornet.nV(), h_triPerVertex);
    triangle_t sum=0;
    for(int i=0; i<hornet.nV(); i++){
        // printf("%d %ld\n", i,outputArray[i]);
        sum+=h_triPerVertex[i];
    }
    free(h_triPerVertex);
    //triangle_t sum=gpu::reduce(hd_triangleData().triPerVertex, hd_triangleData().nv+1);

    return sum;
}


void TriangleCounting2::reset(){
    //printf("Inside reset()\n");
    forAllVertices(hornet, OPERATOR_InitTriangleCounts { triPerVertex });
}

void TriangleCounting2::run() {
    //printf("Inside run()\n");
    forAllAdjUnions(hornet, OPERATOR_AdjIntersectionCountBalanced { triPerVertex }, 1);
    //forAllAdjUnions(hornet, OPERATOR_AdjIntersectionCount { triPerVertex });
}

void TriangleCounting2::run(const int WORK_FACTOR=1){
    forAllAdjUnions(hornet, OPERATOR_AdjIntersectionCountBalanced { triPerVertex }, WORK_FACTOR);
}


void TriangleCounting2::release(){
    //printf("Inside release\n");
    triPerVertex = nullptr;
}

void TriangleCounting2::init(){
    //printf("Inside init. Printing hornet.nV(): %d\n", hornet.nV());
    pool.allocate(&triPerVertex, hornet.nV());
    reset();
}

} // namespace hornets_nest
