#include "hip/hip_runtime.h"
/**
 * @author Federico Busato                                                  <br>
 *         Univerity of Verona, Dept. of Computer Science                   <br>
 *         federico.busato@univr.it
 * @date September, 2017
 * @version v2
 *
 * @copyright Copyright © 2017 Hornet. All rights reserved.
 *
 * @license{<blockquote>
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * * Neither the name of the copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * </blockquote>}
 */
#include "Static/ShortestPath/SSSP.cuh"
#include <Graph/GraphWeight.hpp>

namespace hornets_nest {

const weight_t INF = std::numeric_limits<weight_t>::max();

//------------------------------------------------------------------------------
///////////////
// OPERATORS //
///////////////

struct SSSPOperator {               //deterministic
    weight_t*            d_distances;
    TwoLevelQueue<vid_t> queue;

    OPERATOR(Vertex& vertex, Edge& edge) {
        auto src       = vertex.id();
        auto dst       = edge.dst_id();
        auto weight    = edge.template field<0>();
        auto tentative = d_distances[src] + weight;
        if (atomicMin(d_distances + dst, tentative) > tentative)
            queue.insert(dst);
    }
};
//------------------------------------------------------------------------------
/////////////////
// SSSP //
/////////////////

SSSP::SSSP(HornetGraph& hornet) : StaticAlgorithm(hornet),
                                  queue(hornet, 4.0f),
                                  load_balancing(hornet) {
    pool.allocate(&d_distances, hornet.nV());
    reset();
}

SSSP::~SSSP() {
}

void SSSP::reset() {
    queue.clear();
    auto distances = d_distances;
    forAllnumV(hornet, [=] __device__ (int i){ distances[i] = INF; } );
}

void SSSP::set_parameters(vid_t source) {
    sssp_source = source;
    queue.insert(sssp_source);
    host::copyToDevice(weight_t(0), d_distances + sssp_source);
}

void SSSP::run() {
    int current_level = 0;
    while (queue.size() > 0) {
        forAllEdges(hornet, queue, SSSPOperator { d_distances, queue },
                    load_balancing);
        current_level++;
        queue.swap();
    }
    std::cout<<"Levels : "<<current_level<<"\n";
}

void SSSP::release() {
    d_distances = nullptr;
}

bool SSSP::validate() {
    return true;
}

} // namespace hornets_nest
