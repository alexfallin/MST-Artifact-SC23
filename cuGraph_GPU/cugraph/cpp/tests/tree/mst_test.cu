#include <utilities/base_fixture.hpp>
#include <utilities/high_res_clock.h>
#include <utilities/test_utilities.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/legacy/graph.hpp>

#include <raft/error.hpp>
#include <raft/handle.hpp>

#include <hip/hip_runtime_api.h>

#include <cmath>

#include "../src/converters/legacy/COOtoCSR.cuh"
#include <thrust/device_ptr.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/reduce.h>

#include <sys/time.h>

const int NUM_RUNS = 9;

static double median(double array[], const int n) {
  double median = 0;
  std::sort(array, array + n);
  if (n % 2 == 0) median = (array[(n - 1) / 2] + array[n / 2]) / 2.0;
  else median = array[n / 2];
  return median;
}

int main(int argc, char* argv[]) {

  if (argc != 2) {
    printf("USAGE: %s input_graph\n", argv[0]);
    exit(-1);
  }

  printf("%s\n", argv[1]); //input

  FILE* fpin = fopen(argv[1], "r");

  int m, k, nnz;
  MM_typecode mc;

  cugraph::test::mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz);

  // Allocate memory on host
  std::vector<int> cooRowInd(nnz), cooColInd(nnz);
  std::vector<float> cooVal(nnz), mst(m);

  // Read
  cugraph::test::mm_to_coo<int, float>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL);
  fclose(fpin);

  raft::handle_t handle;

  cugraph::legacy::GraphCOOView<int, int, float> G_coo(
      &cooRowInd[0], &cooColInd[0], &cooVal[0], m, nnz);
  auto G_unique = cugraph::coo_to_csr(G_coo);
  cugraph::legacy::GraphCSRView<int, int, float> G(G_unique->view().offsets,
                                                   G_unique->view().indices,
                                                   G_unique->view().edge_data,
                                                   G_unique->view().number_of_vertices,
                                                   G_unique->view().number_of_edges);

  hipDeviceSynchronize();

  auto mst_edges = cugraph::minimum_spanning_tree<int, int, float>(handle, G);
  hipDeviceSynchronize();

  timeval start, end;

  double runtimes[NUM_RUNS];
  for (int i = 0; i < NUM_RUNS; i++) {
    gettimeofday(&start, NULL);

    mst_edges = cugraph::minimum_spanning_tree<int, int, float>(handle, G);
    hipDeviceSynchronize();

    gettimeofday(&end, NULL);

    runtimes[i] = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  }

  printf("%12.9f s, float_ver\n", median(runtimes, NUM_RUNS));

  auto calculated_mst_weight = thrust::reduce(
      thrust::device_pointer_cast(mst_edges->view().edge_data),
      thrust::device_pointer_cast(mst_edges->view().edge_data) + mst_edges->view().number_of_edges);

//  printf("calc weight: %ld\n", (long)calculated_mst_weight);
//  printf("calc weight: %lx\n", (long)calculated_mst_weight);
//  std::cout << "calculated_mst_weight: " << calculated_mst_weight << std::endl;
//  std::cout << "number_of_MST_edges: " << mst_edges->view().number_of_edges << std::endl;
}
