/*
	Programa: mst_cuda_semGB.c  (Versão 1)
	Descrição: Implementa o Algoritmo para árvore geradora mínima.
	Programadora: Jucele Vasconcellos
	Data: 25/08/2017
	Versão 3: sem o atomicAddD em Calcula_num_zerodiff
	Compilacao:	nvcc -arch sm_30 -o mst_cuda_semGB.exe mst_cuda_semGB.cu
	Execucao:	./st_cuda.exe in/grafo/grafo1000a cuda.out
	
	Entrada de dados: Este programa lê os dados de um grafo no formato
	8
	16
	4 5 0.35
	4 7 0.37
	5 7 0.28
	0 7 0.16
	1 5 0.32
	0 4 0.38
	2 3 0.17
	1 7 0.19
	0 2 0.26
	1 2 0.36
	1 3 0.29
	2 7 0.34
	6 2 0.40
	3 6 0.52
	6 0 0.58
	6 4 0.93

	sendo a primeira linha o número de vértices, a segunda linha o número de arestas
	e as linhas subsequentes as arestas v1 v2 custo
		
	Saída de Dados: Este programa produz um arquivo de saída as arestas que compõem a árvore geradora 
*/

#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>

// Grafo Original
typedef struct {
  unsigned int v, u;
  unsigned int grau_st;
} aresta;

typedef struct {
  int n, m;
  aresta* arestas;
  float* custos;
} grafo;

typedef struct {
  int v1, v2;
} aresta_E;


// Funções e Procedimentos

static void HandleError(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define CHECK_ERROR(err) (HandleError(err, __FILE__, __LINE__))

grafo LeGrafo(char*);
__device__ double atomicAddD(double* address, float val);
__global__ void EncontraMenorAresta1(aresta*, float*, int*, int, int);
__global__ void EncontraMenorAresta2(aresta*, float*, int*, int, int);
__global__ void MarcarArestas_Strut(aresta*, int*, int);
__global__ void Calcula_num_zerodiff(aresta*, float*, int, int*, unsigned int*, unsigned int*, double*);
__global__ void Inicializa_arestasE_C(aresta*, int*, int, int*, aresta_E*, unsigned int*);
__global__ void AtualizaC_1(aresta_E*, int*, int*, int);
__global__ void DefineNovosVU(int*, int, char*, unsigned int*);
__global__ void AtualizaC_3(int*, int, char*);
__global__ void MarcarArestas(aresta*, int*, int, int);

// Função Principal
int main(int argc, char** argv) {
  grafo G;
  double tempoTotal, tempo1, tempo2;
  double tempo1p, tempo2p;
  int* SolutionEdgeSet;
  int SolutionSize, i, it;
  double SolutionVal;
  int num_zerodiff;
  FILE* Arq;

  int dimBloco, dimGrid;
  aresta* d_arestas;
  float* d_custos;
  int* d_SolutionEdgeSet;
  unsigned int* d_SolutionSize, * d_num_zerodiff;
  double* d_SolutionVal;
  int* d_menorAresta;
  int n;


  // Passo 1: Verificação de parâmetros
  // Passo 2: Leitura dos dados do grafo
  // Passo 3: Criação do grafo bipartido correspondente às arestas recebidas
  // Passo 4: Encontra a solução
  // Passo 4.1: Escolher arestas que comporão a strut
  // Passo 4.2: Calcular o num_zerodiff e computar novas componenetes conexas
  // Passo 4.3: Compactar o grafo


  // ==============================================================================
  // Passo 1: Verificação de parâmetros
  // ==============================================================================

  //Verificando os parametros
  if (argc < 3) {
    printf("\nParametros incorretos\n Uso: ./cms_seq.exe <ArqEntrada> <ArqSaida> <dimBloco> <S/N> onde:\n");
    printf("\t <ArqEntrada> (obrigatorio) - Nome do arquivo com as informações do grafo (número de vértices, número de arestas e arestas.\n");
    printf("\t <ArqSaida> (obrigatorio) - Nome do arquivo de saida.\n");
    printf("\t <S ou N> - Mostrar ou não as arestas da MST.\n");
    return 0;
  }
  //Define a dimensão do bloco fixada em 32 threads
  //dimBloco = 32;
  dimBloco = 64;

  // ==============================================================================
  // Passo 2: Leitura dos dados do Grafo G
  // ==============================================================================
//  tempo1p = (double) clock() / CLOCKS_PER_SEC;
  G = LeGrafo(argv[1]);
  n = G.n;
// 	printf("****************************\n");
// 	printf("****** GRAFO ORIGINAL ******\n");
// 	MostraGrafoOriginal(GO);

// 	printf("Grafo de entrada lido\n");

  //Alocação de variável para armazenar solução
  SolutionEdgeSet = (int*) malloc((G.n - 1) * sizeof(int));
  //Aloca memória no device para as arestas da d_SolutionEdgeSet
  CHECK_ERROR(hipMalloc((void**) &d_SolutionEdgeSet, G.n * sizeof(int)));
  SolutionSize = 0;
  SolutionVal = 0;
  //Aloca memória no device para a variável d_SolutionSize
  CHECK_ERROR(hipMalloc((void**) &d_SolutionSize, sizeof(unsigned int)));
  //Inicializa d_SolutionSize com 0
  CHECK_ERROR(hipMemset(d_SolutionSize, 0, sizeof(unsigned int)));
  //Aloca memória no device para a variável d_SolutionVal
  CHECK_ERROR(hipMalloc((void**) &d_SolutionVal, sizeof(double)));
  //Inicializa d_SolutionVal com 0
  CHECK_ERROR(hipMemset(d_SolutionVal, 0, sizeof(double)));
  //Aloca memória no device para a variável d_num_zerodiff
  CHECK_ERROR(hipMalloc((void**) &d_num_zerodiff, sizeof(unsigned int)));

//  tempo2p = (double) clock() / CLOCKS_PER_SEC;
//  	printf("Tempo Passo 2: %lf\n", tempo2p - tempo1p);

  // ==============================================================================
  // Passo 3: Tranfere informações para device e inicializa variáveis
  // ==============================================================================
  //Iniciando contagem do tempo
//	tempo1 = (double) clock( ) / CLOCKS_PER_SEC;
//	tempo1p = (double) clock( ) / CLOCKS_PER_SEC;

  //Aloca memória no device para as arestas do grafo
  CHECK_ERROR(hipMalloc((void**) &d_arestas, G.m * sizeof(aresta)));
  //Copia as arestas do grafo do host para o device
  CHECK_ERROR(hipMemcpy(d_arestas, G.arestas, G.m * sizeof(aresta), hipMemcpyHostToDevice));

  //Aloca memória no device para as arestas do grafo
  CHECK_ERROR(hipMalloc((void**) &d_custos, G.m * sizeof(float)));
  //Copia as arestas do grafo do host para o device
  CHECK_ERROR(hipMemcpy(d_custos, G.custos, G.m * sizeof(float), hipMemcpyHostToDevice));

  //Aloca memória no device para o vetor d_menorAresta
  CHECK_ERROR(hipMalloc((void**) &d_menorAresta, G.n * sizeof(int)));


//	tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
//  	printf("Tempo Passo 3: %lf\n", tempo2p - tempo1p);

  // ==============================================================================
  // Passo 4: Encontra solução
  // ==============================================================================

  timeval start, end;
  gettimeofday(&start, NULL);

  it = 0;
  num_zerodiff = 0;
  while (num_zerodiff != 1) {
    // ==============================================================================
    // Passo 4.1: Escolher arestas que comporão a strut
    // ==============================================================================
//		tempo1p = (double) clock( ) / CLOCKS_PER_SEC;

    //Inicializa d_menorAresta com -1
    CHECK_ERROR(hipMemset(d_menorAresta, -1, G.n * sizeof(int)));

    //Chama kernel para encontrar menorAresta de cada v
    dimGrid = ((G.m - 1) / dimBloco) + 1;
    EncontraMenorAresta1<<<dimGrid, dimBloco>>>(d_arestas, d_custos, d_menorAresta, G.m, G.n);
// 		if(it >= 0)
// 		{
// 			printf("Após EncontraMenorAresta1\n");
// 			int *h_menorAresta;
// 			h_menorAresta = (int *) malloc(G.n * sizeof(int)); 
// 			CHECK_ERROR(cudaMemcpy(h_menorAresta, d_menorAresta, G.n * sizeof(int), cudaMemcpyDeviceToHost));
// 			for(i = 0; i < n; i++)
// 				printf("MenorAresta[%d] = %d\n", i, h_menorAresta[i]);
// 			free(h_menorAresta);
// 		}
    dimGrid = ((G.m - 1) / dimBloco) + 1;
    EncontraMenorAresta2<<<dimGrid, dimBloco>>>(d_arestas, d_custos, d_menorAresta, G.m, G.n);
// 		if(it >= 0)
// 		{
// 			printf("Após EncontraMenorAresta2\n");
// 			int *h_menorAresta;
// 			h_menorAresta = (int *) malloc(G.n * sizeof(int)); 
// 			CHECK_ERROR(cudaMemcpy(h_menorAresta, d_menorAresta, G.n * sizeof(int), cudaMemcpyDeviceToHost));
// 			for(i = 0; i < n; i++)
// 				printf("MenorAresta[%d] = %d\n", i, h_menorAresta[i]);
// 			free(h_menorAresta);
// 		}


    //Chama kernel para marcar arestas da Strut
    dimGrid = ((n - 1) / dimBloco) + 1;
    MarcarArestas_Strut<<<dimGrid, dimBloco>>>(d_arestas, d_menorAresta, n);

//		tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
//  		printf("Tempo Passo 4.1: %lf\n", tempo2p - tempo1p);

    // ==============================================================================
    // Passo 4.2: Calcular o num_zerodiff
    // ==============================================================================
//		tempo1p = (double) clock( ) / CLOCKS_PER_SEC;

    //Inicializa d_num_zerodiff com 0
    CHECK_ERROR(hipMemset(d_num_zerodiff, 0, sizeof(unsigned int)));

    //Chama kernel para calcular num_zerodiff e preencher SolutionEdgeSet
    dimGrid = ((G.m - 1) / dimBloco) + 1;
    Calcula_num_zerodiff<<<dimGrid, dimBloco>>>(d_arestas, d_custos, G.m, d_SolutionEdgeSet, d_SolutionSize, d_num_zerodiff, d_SolutionVal);

    CHECK_ERROR(hipMemcpy(&num_zerodiff, d_num_zerodiff, sizeof(unsigned int), hipMemcpyDeviceToHost));

//		tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
//   		printf("Tempo Passo 4.2: %lf     it = %d     num_zerodiff = %d     SolutionSize = %d\n", tempo2p - tempo1p, it, num_zerodiff, SolutionSize);

    // ==============================================================================
    // Passo 4.3: Compactar o grafo
    // ==============================================================================
    if (num_zerodiff != 1) {
      // ==============================================================================
      // Passo 4.3.1: Computar componenetes conexas
      // ==============================================================================
//			tempo1p = (double) clock( ) / CLOCKS_PER_SEC;

      //Declara variável d_aux
      unsigned int* d_aux;
      //Aloca memória no device para a variável d_aux
      CHECK_ERROR(hipMalloc((void**) &d_aux, sizeof(unsigned int)));
      //Inicializa d_aux com 0
      CHECK_ERROR(hipMemset(d_aux, 0, sizeof(unsigned int)));

      //Declara variável d_arestasE
      aresta_E* d_arestasE;
      //Aloca memória no device para a variável d_arestasE
      CHECK_ERROR(hipMalloc((void**) &d_arestasE, n * sizeof(aresta_E)));

      //Declara variável d_CD
      int* d_C;
      //Aloca memória no device para a variável d_CD
      CHECK_ERROR(hipMalloc((void**) &d_C, n * sizeof(int)));

      //Chama kernel para inicializar d_arestasE e d_C
      dimGrid = ((n - 1) / dimBloco) + 1;
      Inicializa_arestasE_C<<<dimGrid, dimBloco>>>(d_arestas, d_menorAresta, n, d_C, d_arestasE, d_aux);

      int h_fim, * d_fim;
      CHECK_ERROR(hipMalloc((void**) &d_fim, sizeof(int)));
      dimGrid = ((n - 1 - num_zerodiff) / dimBloco) + 1;
      do {
        h_fim = 0;
        CHECK_ERROR(hipMemcpy(d_fim, &h_fim, sizeof(int), hipMemcpyHostToDevice));
        AtualizaC_1<<<dimGrid, dimBloco>>>(d_arestasE, d_C, d_fim, n - num_zerodiff);
        CHECK_ERROR(hipMemcpy(&h_fim, d_fim, sizeof(int), hipMemcpyDeviceToHost));
      } while (h_fim == 1);

      CHECK_ERROR(hipDeviceSynchronize());

      //Declara variável d_aux2
      char* d_aux2;
      //Aloca memória no device para a variável d_aux2
      CHECK_ERROR(hipMalloc((void**) &d_aux2, n * sizeof(char)));
      //Inicializa d_aux2 com 0
      CHECK_ERROR(hipMemset(d_aux2, 0, n * sizeof(char)));
      //Inicializa d_aux com 0
      CHECK_ERROR(hipMemset(d_aux, 0, sizeof(unsigned int)));
      //Chama kernel para atualizar d_C
      dimGrid = ((n - 1) / dimBloco) + 1;
// 			printf("Vou chamar DefineNovosVU para dimGrid = %d  e  dimBloco = %d   com n = %d\n", dimGrid, dimBloco, n);
      DefineNovosVU<<<dimGrid, dimBloco>>>(d_C, n, d_aux2, d_aux);

      CHECK_ERROR(hipDeviceSynchronize());

      //Chama kernel para atualizar d_C
      dimGrid = ((n - 1) / dimBloco) + 1;
      AtualizaC_3<<<dimGrid, dimBloco>>>(d_C, n, d_aux2);

      //Liberando variáveis
      CHECK_ERROR(hipFree(d_aux));
      CHECK_ERROR(hipFree(d_arestasE));
      CHECK_ERROR(hipFree(d_fim));
      CHECK_ERROR(hipFree(d_aux2));

//			tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
//  			printf("Tempo Passo 4.3.1: %lf\n", tempo2p - tempo1p);

      // ==============================================================================
      // Passo 4.3.2: Marcar arestas
      // ==============================================================================
//			tempo1p = (double) clock( ) / CLOCKS_PER_SEC;

      //Marca as arestas para remoção
      dimGrid = ((G.m - 1) / dimBloco) + 1;
      MarcarArestas<<<dimGrid, dimBloco>>>(d_arestas, d_C, G.m, G.n);

      CHECK_ERROR(hipDeviceSynchronize());

      CHECK_ERROR(hipFree(d_C));


// 			aresta *h_arestas;
// 			h_arestas = (aresta *) malloc(G.m * sizeof(aresta)); 
// 			CHECK_ERROR(cudaMemcpy(h_arestas, d_arestas, G.m * sizeof(aresta), cudaMemcpyDeviceToHost));
// 			for(i = 0; i < G.m; i++)
// 				if(h_arestas[i].v != G.n)
// 					printf("Aresta[%d]     v = %d    u = %d\n", i, h_arestas[i].v, h_arestas[i].u);
// 			free(h_arestas);


//			tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
//  			printf("Tempo Passo 4.3.2: %lf\n", tempo2p - tempo1p);		
    }

    it++;
    n = num_zerodiff;
  } // fim while(num_zerodiff != 1)
  gettimeofday(&end, NULL);
  printf("%12.9f s\n", end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0);


  // Copia a Solução para o host
// 	CHECK_ERROR(cudaMemcpy(&SolutionVal, d_SolutionVal, sizeof(double), cudaMemcpyDeviceToHost));
  CHECK_ERROR(hipMemcpy(&SolutionSize, d_SolutionSize, sizeof(unsigned int), hipMemcpyDeviceToHost));
  CHECK_ERROR(hipMemcpy(SolutionEdgeSet, d_SolutionEdgeSet, (G.n - 1) * sizeof(int), hipMemcpyDeviceToHost));

  SolutionVal = 0;
  for (i = 0; i < SolutionSize; i++)
    SolutionVal += G.custos[SolutionEdgeSet[i]];

//	tempo2 = (double) clock( ) / CLOCKS_PER_SEC;
//	tempoTotal = tempo2 - tempo1;
//
//	printf("%lf\n", tempoTotal);

  Arq = fopen(argv[2], "a");
  fprintf(Arq, "\n*** Input file: %s\n", argv[1]);
  fprintf(Arq, "Total Time: %lf\n", tempoTotal);
  fprintf(Arq, "Number of iterations: %d\n", it);
  fprintf(Arq, "SolutionSize: %d\n", SolutionSize);
  fprintf(Arq, "SolutionValue: %f\n", SolutionVal);

  if ((argc == 4) && (argv[3][0] == 'S' || argv[3][0] == 's')) {
    fprintf(Arq, "*** MST formed by %d edges\n", SolutionSize);
    for (i = 0; i < SolutionSize; i++)
      fprintf(Arq, "Edge %d - %d\n", G.arestas[SolutionEdgeSet[i]].v, G.arestas[SolutionEdgeSet[i]].u);
  }
  fclose(Arq);


  // Liberando variávais alocadas no device
  CHECK_ERROR(hipFree(d_SolutionEdgeSet));
  CHECK_ERROR(hipFree(d_SolutionSize));
  CHECK_ERROR(hipFree(d_SolutionVal));
  CHECK_ERROR(hipFree(d_num_zerodiff));
  CHECK_ERROR(hipFree(d_arestas));
  CHECK_ERROR(hipFree(d_custos));
  CHECK_ERROR(hipFree(d_menorAresta));

  // Liberando variávais alocadas no host
  free(G.arestas);
  free(G.custos);
  free(SolutionEdgeSet);

  return 0;

}

// ==============================================================================
// Função LeGrafo:  Lê as informações do Grafo de um arquivo e armazena em uma 
//                  estrutura
// ==============================================================================
grafo LeGrafo(char* Arquivo) {
  int i, aux;
  grafo G;
  FILE* Arq;

  Arq = fopen(Arquivo, "r");

  i = 0;
  fscanf(Arq, "%d", &i);
  G.n = i;

  fscanf(Arq, "%d", &i);
  G.m = i;

  G.arestas = (aresta*) malloc(G.m * sizeof(aresta));
  G.custos = (float*) malloc(G.m * sizeof(float));

  for (i = 0; i < G.m; i++) {
    fscanf(Arq, "%d", &G.arestas[i].u);
    fscanf(Arq, "%d", &G.arestas[i].v);
    if (G.arestas[i].v > G.arestas[i].u) {
      aux = G.arestas[i].v;
      G.arestas[i].v = G.arestas[i].u;
      G.arestas[i].u = aux;
    }
    fscanf(Arq, "%f", &G.custos[i]);
    G.arestas[i].grau_st = 0;
  }

  fclose(Arq);
  return G;
}

__device__ double atomicAddD(double* address, float val) {
  unsigned long long int* address_as_ull = (unsigned long long int*) address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

// ==============================================================================
// Função EncontraMenorAresta1:  Para cada vértice v encontra o id da aresta menor
// ==============================================================================
__global__ void EncontraMenorAresta1(aresta* arestas, float* custos, int* menorAresta, int m, int n) {
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  int x, aux;
  int old;

  if (id < m) {
    if (arestas[id].v != n) {
      x = arestas[id].v;
// 			printf("1.1 Aresta %d   v = %d   u = %d  custo = %f    menorAresta[%d] = %d\n", id, arestas[id].v, arestas[id].u, custos[id], x, menorAresta[x]);
      if ((menorAresta[x] == -1) ||
          (custos[menorAresta[x]] > custos[id]) ||
          ((custos[menorAresta[x]] == custos[id]) && (menorAresta[x] > id))) {
        aux = -1;
        // atomicCAS = atomic Compare And Swap
        // lê o conteúdo endereçado por menorAresta[x] e o armazena em old.
        // computa (old == aux ? meu_id: old)
        // ou seja se (old == aux) então menorAresta[x] = id
        //         senão menorAresta[x] = old
        // retorna old
        old = atomicCAS(&menorAresta[x], aux, id);
        if (old != aux) {
          while (((custos[old] > custos[id]) || ((custos[old] == custos[id]) && (old > id))) && (old != aux)) {
            aux = atomicCAS(&menorAresta[x], old, id);
            if (old != aux) {
              old = aux;
              aux = -1;
            }
          }// fim while((old > id) && (old != aux))
        } // fim if(old_id != aux)
// 				printf("1.2 Aresta %d   v = %d   u = %d  custo = %f    menorAresta[%d] = %d\n", id, arestas[id].v, arestas[id].u, custos[id], x, menorAresta[x]);
      } // fim if((menorAresta[x] == -1) || (menorAresta[x] > id))
    } // fim if(arestas[id].v != n)
  } //fim if( id < m )
}

// ==============================================================================
// Função EncontraMenorAresta2:  Para cada vértice v encontra o id da aresta menor
// ==============================================================================
__global__ void EncontraMenorAresta2(aresta* arestas, float* custos, int* menorAresta, int m, int n) {
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  int x, aux;
  int old;

  if (id < m) {
    if (arestas[id].v != n) {
      x = arestas[id].u;
// 			printf("2.1 Aresta %d   v = %d   u = %d  custo = %f    menorAresta[%d] = %d\n", id, arestas[id].v, arestas[id].u, custos[id], x, menorAresta[x]);
      if ((menorAresta[x] == -1) ||
          (custos[menorAresta[x]] > custos[id]) ||
          ((custos[menorAresta[x]] == custos[id]) && (menorAresta[x] > id))) {
        aux = -1;
        // atomicCAS = atomic Compare And Swap
        // lê o conteúdo endereçado por menorAresta[x] e o armazena em old.
        // computa (old == aux ? meu_id: old)
        // ou seja se (old == aux) então menorAresta[x] = id
        //         senão menorAresta[x] = old
        // retorna old
        old = atomicCAS(&menorAresta[x], aux, id);
        if (old != aux) {
          while (((custos[old] > custos[id]) || ((custos[old] == custos[id]) && (old > id))) && (old != aux)) {
            aux = atomicCAS(&menorAresta[x], old, id);
            if (old != aux) {
              old = aux;
              aux = -1;
            }
          }// fim while((old > id) && (old != aux))
        } // fim if(old_id != aux)
// 				printf("2.2 Aresta %d   v = %d   u = %d  custo = %f    menorAresta[%d] = %d\n", id, arestas[id].v, arestas[id].u, custos[id], x, menorAresta[x]);
      } // if((menorAresta[x] == -1) || (menorAresta[x] > id))
    } // fim if(arestas[id].v != n)
  } //fim if( id < m )
}

// ==============================================================================
// Função MarcarArestas_Strut:  Para cada vértice v marca aresta pertencente a strut
// ==============================================================================
__global__ void MarcarArestas_Strut(aresta* d_arestas, int* d_menorAresta, int n) {
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  if (id < n) {
    atomicInc(&d_arestas[d_menorAresta[id]].grau_st, UINT_MAX);
// 		printf("MenorAresta[%d] = %d\n", id, d_menorAresta[id]);
  }
}

// ==============================================================================
// Função Calcula_num_zerodiff:  Calcula o número de vértices zero diferença e
//                               preenche o vetor d_SolutionEdgeSet
// ==============================================================================
__global__ void Calcula_num_zerodiff(aresta* arestas, float* custos, int m, int* SolutionEdgeSet, unsigned int* SolutionSize, unsigned int* num_zerodiff, double* SolutionVal) {
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  int pos;

  if (id < m) {
    if (arestas[id].grau_st > 0) {
      pos = atomicInc(&SolutionSize[0], UINT_MAX);
      SolutionEdgeSet[pos] = id;
// 			atomicAddD(&SolutionVal[0], custos[id]);
      if (arestas[id].grau_st == 2) {
        pos = atomicInc(&num_zerodiff[0], UINT_MAX);
// 				printf("num_zerodiff = %d\n", num_zerodiff[0]);
      }
    }
  }
}

// ==============================================================================
// Função Inicializa_arestasE_CD:  Inicializa as variáveis CD, arestasE e arestasE_size
// ==============================================================================
__global__ void Inicializa_arestasE_C(aresta* arestas, int* menorAresta, int n, int* C, aresta_E* arestasE, unsigned int* arestasE_size) {
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  int pos;

  if (id < n) {
    if ((arestas[menorAresta[id]].grau_st == 1) || (((arestas[menorAresta[id]].grau_st == 2)) && (id == arestas[menorAresta[id]].v))) {
      pos = atomicInc(&arestasE_size[0], UINT_MAX);
      arestasE[pos].v1 = arestas[menorAresta[id]].v;
      arestasE[pos].v2 = arestas[menorAresta[id]].u;
// 			printf("Thread %d    arestasE[%d]   v1 = %d   v2 = %d\n", id, pos, arestasE[pos].v1, arestasE[pos].v2);
    }
    C[id] = id;
  }
}

// ==============================================================================
// Função AtualizaC_1: Atualiza vetor C para definição das componentes conexas
// ==============================================================================
__global__ void AtualizaC_1(aresta_E* arestasE, int* C, int* m, int n) {
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  int c1, c2, v1, v2;

  if (id < n) {
    v1 = arestasE[id].v1;
    v2 = arestasE[id].v2;
    c1 = C[v1];
    c2 = C[v2];
    if (c1 < c2) {
      atomicMin(&C[v2], c1);
      m[0] = 1;
    } else if (c2 < c1) {
      atomicMin(&C[v1], c2);
      m[0] = 1;
    }
  }
}

// ==============================================================================
// Função DefineNovosVU: Atualiza vetor C
// ==============================================================================
__global__ void DefineNovosVU(int* C, int n, char* marcador, unsigned int* num_comp) {
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  int pos;

  if (id < n) {
    if (C[id] == id) {
      pos = atomicInc(&num_comp[0], UINT_MAX);
      C[id] = pos;
      marcador[id] = 1;
      //printf("C2[%d] = %d\n", id, C[id]);
    }
  }
}

// ==============================================================================
// Função AtualizaC_3: Atualiza vetor C
// ==============================================================================
__global__ void AtualizaC_3(int* C, int n, char* marcador) {
  int id = threadIdx.x + blockDim.x * blockIdx.x;

  if (id < n) {
    if (marcador[id] == 0) {
      C[id] = C[C[id]];
      //printf("C3[%d] = %d\n", id, C[id]);
    }
  }
}

// ==============================================================================
// Função MarcarArestas:  Marca as arestas do grafo a serem removidas
// ==============================================================================
__global__ void MarcarArestas(aresta* arestas, int* C, int m, int n) {
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  int x, y;

  if (id < m) {
    if (arestas[id].v != n) {
      x = C[arestas[id].v];
      y = C[arestas[id].u];

      if (x != y) {
        arestas[id].v = x;
        arestas[id].u = y;
      } else {
        arestas[id].v = n;
        arestas[id].u = n;
      }
      arestas[id].grau_st = 0;
    }
  } //fim if( id < m )
}


