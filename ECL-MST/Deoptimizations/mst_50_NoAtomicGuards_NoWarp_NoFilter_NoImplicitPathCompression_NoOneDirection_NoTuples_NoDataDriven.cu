#include "hip/hip_runtime.h"
/*
ECL-MST: This code computes the minimum spanning forest of a given input graph.
 
Copyright (c) 2023, Alex Fallin and Martin Burtscher
 
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
 
1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.
 
2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.
 
3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.
 
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 
URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/ECL-MST/.
 
Publication: This work is described in detail in the following paper.
Alex Fallin, Andres Gonzales, Jarim Seo, Randy Cornell, and Martin Burtscher. "A High-Performance MST Implementation for GPUs." Proceedings of the 2023 ACM/IEEE International Conference for High Performance Computing, Networking, Storage, and Analysis. November 2023.
*/


#include <climits>
#include <algorithm>
#include <tuple>
#include <vector>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "ECLgraph.h"

static const int Device = 0;
static const int ThreadsPerBlock = 512;
static const int RUNS = 9;

typedef unsigned long long ull;

static inline int serial_find(const int idx, int* const parent)
{
  int curr = parent[idx];
  if (curr != idx) {
    int next, prev = idx;
    while (curr != (next = parent[curr])) {
      parent[prev] = next;
      prev = curr;
      curr = next;
    }
  }
  return curr;
}

static inline void serial_join(const int a, const int b, int* const parent)
{
  const int arep = serial_find(a, parent);
  const int brep = serial_find(b, parent);
  if (arep > brep) {  // improves locality
    parent[brep] = arep;
  } else {
    parent[arep] = brep;
  }
}

static bool* cpuMST(const ECLgraph& g)
{
  bool* const inMST = new bool [g.edges];
  int* const parent = new int [g.nodes];

  timeval start, end;
  gettimeofday(&start, NULL);

  std::fill(inMST, inMST + g.edges, false);
  for (int i = 0; i < g.nodes; i++) parent[i] = i;

  std::vector<std::tuple<int, int, int, int>> list;  // <weight, edge index, from node, to node>
  for (int i = 0; i < g.nodes; i++) {
    for (int j = g.nindex[i]; j < g.nindex[i + 1]; j++) {
      const int n = g.nlist[j];
      if (n > i) {  // only one direction
        list.push_back(std::make_tuple(g.eweight[j], j, i, n));
      }
    }
  }
  std::sort(list.begin(), list.end());

  int count = g.nodes - 1;
  for (int pos = 0; pos < list.size(); pos++) {
    const int a = std::get<2>(list[pos]);
    const int b = std::get<3>(list[pos]);
    const int arep = serial_find(a, parent);
    const int brep = serial_find(b, parent);
    if (arep != brep) {
      const int j = std::get<1>(list[pos]);
      inMST[j] = true;
      serial_join(arep, brep, parent);
      count--;
      if (count == 0) break;
    }
  }

  gettimeofday(&end, NULL);
//  printf("Host: %12.9f s\n", end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0);

  delete [] parent;
  return inMST;
}

static inline __device__ int find(const int idx, volatile int* const __restrict__ parent) {
  int curr = parent[idx];
  if (curr != idx) {
    int next, prev = idx;
    while (curr != (next = parent[curr])) {
      parent[prev] = next;
      prev = curr;
      curr = next;
    }
  }
  return curr;
}

static inline __device__ void join(int arep, int brep, int* const __restrict__ parent)
{
  int mrep;
  do {
    mrep = max(arep, brep);
    arep = min(arep, brep);
  } while ((brep = atomicCAS(&parent[mrep], mrep, arep)) != mrep);
}

static __global__ void initPM(const int nodes, int* const __restrict__ parent, ull* const __restrict__ minv)
{
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < nodes) {
    parent[v] = v;
    minv[v] = ULONG_MAX;
  }
}

template <bool first>
static __global__ void initWL(const int nodes, const int* const __restrict__ nindex, int* const __restrict__ slist, const int* const __restrict__ nlist, const int* const __restrict__ eweight, ull* const __restrict__ minv, int* const __restrict__ parent, const int threshold)
{
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  int beg, end;
  if (v < nodes) {
    beg = nindex[v];
    end = nindex[v + 1];
    for (int j = beg; j < end; j++) {
      slist[j] = v;
    }
  }
}

static __global__ void kernel1(const int wl1size, bool* const __restrict__ go_again, int* const __restrict__ parent, ull* const __restrict__ minv, const int* const __restrict__ slist, const int* const __restrict__ nlist, const int* const __restrict__ eweight)
{
  const int j = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (j < wl1size) {
    const int arep = find(slist[j], parent);
    const int brep = find(nlist[j], parent);
    if (arep != brep) {
      *go_again = true;
      const ull val = (((ull)eweight[j]) << 32) | j;
      atomicMin(&minv[arep], val);
      atomicMin(&minv[brep], val);
    }
  }
}

static __global__ void kernel2(const int wlsize, int* const __restrict__ parent, ull* const __restrict__ minv, bool* const __restrict__ inMST, const int* const __restrict__ slist, const int* const __restrict__ nlist, const int* const __restrict__ eweight)
{
  const int j = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (j < wlsize) {
    const int arep = find(slist[j], parent);
    const int brep = find(nlist[j], parent);
    if (arep != brep) {
      const ull val = (((ull)eweight[j]) << 32) | j;
      if ((val == minv[arep]) || (val == minv[brep])) {
        join(arep, brep, parent);
        inMST[j] = true;
      }
    }
  }
}

static __global__ void kernel3(const int wlsize, int* const __restrict__ parent, volatile ull* const __restrict__ minv, const int* const __restrict__ slist, const int* const __restrict__ nlist)
{
  const int j = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (j < wlsize) {
    const int arep = find(slist[j], parent);
    const int brep = find(nlist[j], parent);
    if (arep != brep) {
      minv[arep] = ULONG_MAX;
      minv[brep] = ULONG_MAX;
    }
  }
}

static void CheckCuda(const int line)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d on line %d: %s\n", e, line, hipGetErrorString(e));
    exit(-1);
  }
}

template <bool filter>
static bool* gpuMST(const ECLgraph& g, const int threshold)
{
  bool* d_inMST = NULL;
  hipMalloc((void**)&d_inMST, g.edges * sizeof(bool));
  bool* const inMST = new bool [g.edges];

  int* d_parent = NULL;
  hipMalloc((void**)&d_parent, g.nodes * sizeof(int));

  ull* d_minv = NULL;
  hipMalloc((void**)&d_minv, g.nodes * sizeof(ull));

  bool* d_go_again = NULL;
  hipMalloc((void**)&d_go_again, sizeof(bool));

  int* d_nindex = NULL;
  hipMalloc((void**)&d_nindex, (g.nodes + 1) * sizeof(int));
  hipMemcpy(d_nindex, g.nindex, (g.nodes + 1) * sizeof(int), hipMemcpyHostToDevice);

  int* d_nlist = NULL;
  hipMalloc((void**)&d_nlist, g.edges * sizeof(int));
  hipMemcpy(d_nlist, g.nlist, g.edges * sizeof(int), hipMemcpyHostToDevice);

  int* d_slist = NULL;
  hipMalloc((void**)&d_slist, g.edges * sizeof(int));

  int* d_eweight = NULL;
  hipMalloc((void**)&d_eweight, g.edges * sizeof(int));
  hipMemcpy(d_eweight, g.eweight, g.edges * sizeof(int), hipMemcpyHostToDevice);

  CheckCuda(__LINE__);

  timeval start, end;
  gettimeofday(&start, NULL);

  int blocks = (g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock;
  initPM<<<blocks, ThreadsPerBlock>>>(g.nodes, d_parent, d_minv);
  hipMemset(d_inMST, 0, g.edges * sizeof(bool));

  initWL<true><<<blocks, ThreadsPerBlock>>>(g.nodes, d_nindex, d_slist, d_nlist, d_eweight, d_minv, d_parent, threshold);

  blocks = (g.edges + ThreadsPerBlock - 1) / ThreadsPerBlock;
  bool go_again = (g.edges > 0);
  while (go_again) {
    hipMemset(d_go_again, false, sizeof(bool));
    kernel1<<<blocks, ThreadsPerBlock>>>(g.edges, d_go_again, d_parent, d_minv, d_slist, d_nlist, d_eweight);
    hipMemcpy(&go_again, d_go_again, sizeof(bool), hipMemcpyDeviceToHost);
    if (go_again) {
      kernel2<<<blocks, ThreadsPerBlock>>>(g.edges, d_parent, d_minv, d_inMST, d_slist, d_nlist, d_eweight);
      kernel3<<<blocks, ThreadsPerBlock>>>(g.edges, d_parent, d_minv, d_slist, d_nlist);
    }
  }

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
//  printf("Dev%c: %12.9f s\n", filter ? '+' : '-', end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0);
  printf("%12.9f\n", end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0);

  hipMemcpy(inMST, d_inMST, g.edges * sizeof(bool), hipMemcpyDeviceToHost);

  hipFree(d_inMST);
  hipFree(d_parent);
  hipFree(d_minv);
  hipFree(d_go_again);
  hipFree(d_nindex);
  hipFree(d_slist);
  hipFree(d_nlist);
  hipFree(d_eweight);

  CheckCuda(__LINE__);

  return inMST;
}

static void verify(const ECLgraph& g, const bool* const cpuMSTedges, const bool* const gpuMSTedges)
{
  int bothMST = 0, neitherMST = 0, onlyCpuMST = 0, onlyGpuMST = 0;
  ull cpuMSTweight = 0, gpuMSTweight = 0;

  for (int j = 0; j < g.edges; j++) {
    const bool inCpuMST = cpuMSTedges[j];
    const bool inGpuMST = gpuMSTedges[j];
    if (inCpuMST && inGpuMST) bothMST++;
    if (!inCpuMST && !inGpuMST) neitherMST++;
    if (!inCpuMST && inGpuMST) onlyGpuMST++;
    if (inCpuMST && !inGpuMST) onlyCpuMST++;
    if (gpuMSTedges[j]) gpuMSTweight += g.eweight[j];
    if (cpuMSTedges[j]) cpuMSTweight += g.eweight[j];
  }

  if ((gpuMSTweight != cpuMSTweight) || (onlyGpuMST != 0) || (onlyCpuMST != 0)) {
    printf("ERROR: results differ!\n\n");
  } else {
//    printf("all good\n\n");
  }
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("USAGE: %s input_graph\n", argv[0]);
    exit(-1);
  }
  ECLgraph g = readECLgraph(argv[1]);

  printf("input: %s\n", argv[1]);

  // assign weights if needed
  if (g.eweight == NULL) {
    g.eweight = new int [g.edges];
    for (int i = 0; i < g.nodes; i++) {
      for (int j = g.nindex[i]; j < g.nindex[i + 1]; j++) {
        const int k = g.nlist[j];
        g.eweight[j] = 1 + ((i * k) % g.nodes);
        if (g.eweight[j] < 0) g.eweight[j] = -g.eweight[j];
      }
    }
  }

  hipSetDevice(Device);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, Device);
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
    fprintf(stderr, "ERROR: there is no CUDA capable device\n\n");
    exit(-1);
  }

  // run GPU code
  bool* gpuMSTedges;
  for (int i = 0; i < RUNS; i++) {
    gpuMSTedges = gpuMST<false>(g, INT_MAX);
  }

  // run CPU code and compare result
  bool* cpuMSTedges = cpuMST(g);
  verify(g, cpuMSTedges, gpuMSTedges);

  freeECLgraph(g);
  delete [] gpuMSTedges;
  delete [] cpuMSTedges;
  return 0;
}
